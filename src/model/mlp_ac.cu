#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CEIL_DIV(a, b) (((a) + (b) - 1) / (b))

#define CHECK_CUDA(call)                                                       \
  {                                                                            \
    hipError_t err = (call);                                                  \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,        \
              hipGetErrorString(err));                                        \
      exit(1);                                                                 \
    }                                                                          \
  }

#define WARP_SIZE 32
#define BLOCK_DIM 256
#define TILE_SIZE 16

// namespace cg = cooperative_groups;

// 优化后的前向传播核函数 (支持混合精度)
__global__ void optimized_linear_forward_kernel(
    const float *__restrict__ input, const float *__restrict__ weights,
    const float *__restrict__ bias, float *__restrict__ output, int input_dim,
    int output_dim, int batch_size) {

  extern __shared__ float shared_mem[];
  float *sh_input = shared_mem;
  float *sh_weights = &shared_mem[TILE_SIZE * TILE_SIZE];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float sum = 0.0f;

  for (int tile = 0; tile < (input_dim + TILE_SIZE - 1) / TILE_SIZE; tile++) {
    int input_col = tile * TILE_SIZE + threadIdx.x;
    int weight_row = tile * TILE_SIZE + threadIdx.y;

    // 加载输入瓦片
    if (row < batch_size && input_col < input_dim) {
      sh_input[threadIdx.y * TILE_SIZE + threadIdx.x] =
          input[row * input_dim + input_col];
    } else {
      sh_input[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
    }

    // 加载权重瓦片
    if (col < output_dim && weight_row < input_dim) {
      sh_weights[threadIdx.y * TILE_SIZE + threadIdx.x] =
          weights[col * input_dim + weight_row];
    } else {
      sh_weights[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
    }

    __syncthreads();

    // 计算部分和
    for (int k = 0; k < TILE_SIZE; k++) {
      sum += sh_input[threadIdx.y * TILE_SIZE + k] *
             sh_weights[k * TILE_SIZE + threadIdx.x];
    }

    __syncthreads();
  }

  if (row < batch_size && col < output_dim) {
    // 添加偏置并存储结果
    output[row * output_dim + col] = sum + bias[col];
  }
}

// 优化的激活前向传播 (tanh)
__global__ void fc_tanh_forward_kernel(const float *__restrict__ input,
                                       const float *__restrict__ weights,
                                       const float *__restrict__ bias,
                                       float *__restrict__ output,
                                       int input_dim, int output_dim,
                                       int batch_size) {

  // 与优化后的线性前向传播相同，但最后应用tanh
  extern __shared__ float shared_mem[];
  float *sh_input = shared_mem;
  float *sh_weights = &shared_mem[TILE_SIZE * TILE_SIZE];

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float sum = 0.0f;

  for (int tile = 0; tile < (input_dim + TILE_SIZE - 1) / TILE_SIZE; tile++) {
    int input_col = tile * TILE_SIZE + threadIdx.x;
    int weight_row = tile * TILE_SIZE + threadIdx.y;

    if (row < batch_size && input_col < input_dim) {
      sh_input[threadIdx.y * TILE_SIZE + threadIdx.x] =
          input[row * input_dim + input_col];
    } else {
      sh_input[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
    }

    if (col < output_dim && weight_row < input_dim) {
      sh_weights[threadIdx.y * TILE_SIZE + threadIdx.x] =
          weights[col * input_dim + weight_row];
    } else {
      sh_weights[threadIdx.y * TILE_SIZE + threadIdx.x] = 0.0f;
    }

    __syncthreads();

    for (int k = 0; k < TILE_SIZE; k++) {
      sum += sh_input[threadIdx.y * TILE_SIZE + k] *
             sh_weights[k * TILE_SIZE + threadIdx.x];
    }

    __syncthreads();
  }

  if (row < batch_size && col < output_dim) {
    output[row * output_dim + col] = tanhf(sum + bias[col]);
  }
}

// 优化的权重梯度计算 (使用共享内存归约)
__global__ void
optimized_weight_grad_kernel(const float *__restrict__ input,
                             const float *__restrict__ grad_output,
                             float *__restrict__ grad_weights, int input_dim,
                             int output_dim, int batch_size) {

  extern __shared__ float shared_mem[];
  float *sh_grad = shared_mem;

  int i = blockIdx.x; // 输出维度索引
  int j = blockIdx.y; // 输入维度索引
  int tid = threadIdx.x;

  float sum = 0.0f;

  // 每个线程处理batch_size/BLOCK_DIM个样本
  for (int b = tid; b < batch_size; b += blockDim.x) {
    sum += input[b * input_dim + j] * grad_output[b * output_dim + i];
  }

  sh_grad[tid] = sum;
  __syncthreads();

  // 块内归约
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      sh_grad[tid] += sh_grad[tid + stride];
    }
    __syncthreads();
  }

  // 使用原子操作更新全局内存
  if (tid == 0) {
    atomicAdd(&grad_weights[i * input_dim + j], sh_grad[0]);
  }
}

// 优化的偏置梯度计算
__global__ void
optimized_bias_grad_kernel(const float *__restrict__ grad_output,
                           float *__restrict__ grad_bias, int output_dim,
                           int batch_size) {

  extern __shared__ float shared_mem[];
  float *sh_grad = shared_mem;

  int i = blockIdx.x; // 输出维度索引
  int tid = threadIdx.x;

  float sum = 0.0f;

  // 每个线程处理batch_size/BLOCK_DIM个样本
  for (int b = tid; b < batch_size; b += blockDim.x) {
    sum += grad_output[b * output_dim + i];
  }

  sh_grad[tid] = sum;
  __syncthreads();

  // 块内归约
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      sh_grad[tid] += sh_grad[tid + stride];
    }
    __syncthreads();
  }

  // 使用原子操作更新全局内存
  if (tid == 0) {
    atomicAdd(&grad_bias[i], sh_grad[0]);
  }
}

// 计算隐藏层梯度 (用于反向传播)
__global__ void compute_hidden_grad_kernel(
    const float *__restrict__ grad_output, const float *__restrict__ weights,
    float *__restrict__ grad_hidden, int in_dim, int out_dim, int batch_size) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= batch_size * in_dim)
    return;

  int b = idx / in_dim;
  int j = idx % in_dim;
  float sum = 0.0f;

  for (int i = 0; i < out_dim; ++i) {
    sum += grad_output[b * out_dim + i] * weights[i * in_dim + j];
  }

  grad_hidden[idx] = sum;
}

// 计算带激活的隐藏层梯度 (tanh)
__global__ void compute_tanh_hidden_grad_kernel(
    const float *__restrict__ grad_output, const float *__restrict__ weights,
    const float *__restrict__ hidden, // 前向传播的隐藏层输出
    float *__restrict__ grad_hidden, int in_dim, int out_dim, int batch_size) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= batch_size * in_dim)
    return;

  int b = idx / in_dim;
  int j = idx % in_dim;
  float sum = 0.0f;

  // 计算权重梯度部分
  for (int i = 0; i < out_dim; ++i) {
    sum += grad_output[b * out_dim + i] * weights[i * in_dim + j];
  }

  // 应用tanh导数 (1 - tanh^2)
  float tanh_val = hidden[b * in_dim + j];
  float tanh_derivative = 1.0f - tanh_val * tanh_val;

  grad_hidden[idx] = sum * tanh_derivative;
}

// 共享层反向传播 (合并Actor和Critic的梯度)
__global__ void optimized_shared_backward_kernel(
    const float *__restrict__ input, const float *__restrict__ grad_actor,
    const float *__restrict__ grad_critic, float *__restrict__ grad_weights,
    float *__restrict__ grad_bias, int input_dim, int hidden_dim,
    int batch_size) {

  // 使用二维块布局处理权重梯度
  int i = blockIdx.x * blockDim.x + threadIdx.x; // hidden_dim
  int j = blockIdx.y * blockDim.y + threadIdx.y; // input_dim

  if (i < hidden_dim && j < input_dim) {
    float sum = 0.0f;
    for (int b = 0; b < batch_size; b++) {
      float delta =
          grad_actor[b * hidden_dim + i] + grad_critic[b * hidden_dim + i];
      sum += input[b * input_dim + j] * delta;
    }
    grad_weights[i * input_dim + j] = sum;
  }

  // 使用单独的块处理偏置梯度
  if (blockIdx.z == 0 && i < hidden_dim && j == 0) {
    float sum = 0.0f;
    for (int b = 0; b < batch_size; b++) {
      float delta =
          grad_actor[b * hidden_dim + i] + grad_critic[b * hidden_dim + i];
      sum += delta;
    }
    grad_bias[i] = sum;
  }
}

// 中间结果结构体
struct IntermediateData {
  float *d_shared_output;
  float *d_actor_hidden;
  float *d_critic_hidden;
  float *d_actor_fc2_output;
  float *d_critic_fc2_output;
  float *d_actor_output;
  float *d_critic_output;
};

// 内存管理助手函数
void cuda_alloc_mem(IntermediateData *data, int batch_size, int input_dim,
                    int hidden_dim, int output_dim) {
  CHECK_CUDA(hipMalloc(&data->d_shared_output,
                        batch_size * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_actor_hidden,
                        batch_size * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_critic_hidden,
                        batch_size * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_actor_fc2_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_critic_fc2_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_actor_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&data->d_critic_output,
                        batch_size * output_dim * sizeof(float)));
}

void cuda_free_mem(IntermediateData *data) {
  CHECK_CUDA(hipFree(data->d_shared_output));
  CHECK_CUDA(hipFree(data->d_actor_hidden));
  CHECK_CUDA(hipFree(data->d_critic_hidden));
  CHECK_CUDA(hipFree(data->d_actor_fc2_output));
  CHECK_CUDA(hipFree(data->d_critic_fc2_output));
  CHECK_CUDA(hipFree(data->d_actor_output));
  CHECK_CUDA(hipFree(data->d_critic_output));
}

// 优化后的前向传播
extern "C" {
void cuda_forward(const float *input, int batch_size, int input_dim,
                  int hidden_dim, int output_dim, const float *shared_w,
                  const float *shared_b, const float *actor_fc1_w,
                  const float *actor_fc1_b, const float *actor_fc2_w,
                  const float *actor_fc2_b, const float *actor_head_w,
                  const float *actor_head_b, const float *critic_fc1_w,
                  const float *critic_fc1_b, const float *critic_fc2_w,
                  const float *critic_fc2_b, const float *critic_head_w,
                  const float *critic_head_b, float *actor_output,
                  float *critic_output, IntermediateData *intermediates) {

  // 分配中间结果内存
  cuda_alloc_mem(intermediates, batch_size, input_dim, hidden_dim, output_dim);

  // 设备指针
  float *d_input, *d_shared_w, *d_shared_b;
  float *d_actor_fc1_w, *d_actor_fc1_b, *d_actor_fc2_w, *d_actor_fc2_b;
  float *d_actor_head_w, *d_actor_head_b;
  float *d_critic_fc1_w, *d_critic_fc1_b, *d_critic_fc2_w, *d_critic_fc2_b;
  float *d_critic_head_w, *d_critic_head_b;

  // 分配设备内存
  CHECK_CUDA(hipMalloc(&d_input, batch_size * input_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_shared_w, hidden_dim * input_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_shared_b, hidden_dim * sizeof(float)));

  CHECK_CUDA(
      hipMalloc(&d_actor_fc1_w, hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_actor_fc1_b, hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_actor_fc2_w, output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_actor_fc2_b, output_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_actor_head_w, output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_actor_head_b, output_dim * sizeof(float)));

  CHECK_CUDA(
      hipMalloc(&d_critic_fc1_w, hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_critic_fc1_b, hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_critic_fc2_w, output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_critic_fc2_b, output_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_critic_head_w, output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_critic_head_b, output_dim * sizeof(float)));

  // 拷贝数据到设备
  CHECK_CUDA(hipMemcpy(d_input, input, batch_size * input_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_shared_w, shared_w,
                        hidden_dim * input_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_shared_b, shared_b, hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(d_actor_fc1_w, actor_fc1_w,
                        hidden_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_fc1_b, actor_fc1_b, hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_fc2_w, actor_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_fc2_b, actor_fc2_b, output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_head_w, actor_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_head_b, actor_head_b,
                        output_dim * sizeof(float), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMemcpy(d_critic_fc1_w, critic_fc1_w,
                        hidden_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_fc1_b, critic_fc1_b,
                        hidden_dim * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_fc2_w, critic_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_fc2_b, critic_fc2_b,
                        output_dim * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_head_w, critic_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_head_b, critic_head_b,
                        output_dim * sizeof(float), hipMemcpyHostToDevice));

  // 创建CUDA流用于并行计算
  hipStream_t actor_stream, critic_stream;
  CHECK_CUDA(hipStreamCreate(&actor_stream));
  CHECK_CUDA(hipStreamCreate(&critic_stream));

  // 计算核函数配置
  dim3 block(TILE_SIZE, TILE_SIZE);
  size_t shared_mem_size = 2 * TILE_SIZE * TILE_SIZE * sizeof(float);

  // 执行共享层前向传播
  dim3 grid_shared((hidden_dim + TILE_SIZE - 1) / TILE_SIZE,
                   (batch_size + TILE_SIZE - 1) / TILE_SIZE);
  optimized_linear_forward_kernel<<<grid_shared, block, shared_mem_size>>>(
      d_input, d_shared_w, d_shared_b, intermediates->d_shared_output,
      input_dim, hidden_dim, batch_size);

  // 并行执行Actor和Critic的前向传播
  // Actor路径
  dim3 grid_actor_fc1((hidden_dim + TILE_SIZE - 1) / TILE_SIZE,
                      (batch_size + TILE_SIZE - 1) / TILE_SIZE);
  fc_tanh_forward_kernel<<<grid_actor_fc1, block, shared_mem_size,
                           actor_stream>>>(
      intermediates->d_shared_output, d_actor_fc1_w, d_actor_fc1_b,
      intermediates->d_actor_hidden, hidden_dim, hidden_dim, batch_size);

  dim3 grid_actor_fc2((output_dim + TILE_SIZE - 1) / TILE_SIZE,
                      (batch_size + TILE_SIZE - 1) / TILE_SIZE);
  fc_tanh_forward_kernel<<<grid_actor_fc2, block, shared_mem_size,
                           actor_stream>>>(
      intermediates->d_actor_hidden, d_actor_fc2_w, d_actor_fc2_b,
      intermediates->d_actor_fc2_output, hidden_dim, output_dim, batch_size);

  // Actor Head层
  optimized_linear_forward_kernel<<<grid_actor_fc2, block, shared_mem_size,
                                    actor_stream>>>(
      intermediates->d_actor_fc2_output, d_actor_head_w, d_actor_head_b,
      intermediates->d_actor_output, output_dim, output_dim, batch_size);

  // Critic路径
  dim3 grid_critic_fc1((hidden_dim + TILE_SIZE - 1) / TILE_SIZE,
                       (batch_size + TILE_SIZE - 1) / TILE_SIZE);
  fc_tanh_forward_kernel<<<grid_critic_fc1, block, shared_mem_size,
                           critic_stream>>>(
      intermediates->d_shared_output, d_critic_fc1_w, d_critic_fc1_b,
      intermediates->d_critic_hidden, hidden_dim, hidden_dim, batch_size);

  dim3 grid_critic_fc2((output_dim + TILE_SIZE - 1) / TILE_SIZE,
                       (batch_size + TILE_SIZE - 1) / TILE_SIZE);
  fc_tanh_forward_kernel<<<grid_critic_fc2, block, shared_mem_size,
                           critic_stream>>>(
      intermediates->d_critic_hidden, d_critic_fc2_w, d_critic_fc2_b,
      intermediates->d_critic_fc2_output, hidden_dim, output_dim, batch_size);

  // Critic Head层
  optimized_linear_forward_kernel<<<grid_critic_fc2, block, shared_mem_size,
                                    critic_stream>>>(
      intermediates->d_critic_fc2_output, d_critic_head_w, d_critic_head_b,
      intermediates->d_critic_output, output_dim, output_dim, batch_size);

  // 等待两个流完成
  CHECK_CUDA(hipStreamSynchronize(actor_stream));
  CHECK_CUDA(hipStreamSynchronize(critic_stream));

  // 拷贝结果回主机
  CHECK_CUDA(hipMemcpy(actor_output, intermediates->d_actor_output,
                        batch_size * output_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(critic_output, intermediates->d_critic_output,
                        batch_size * output_dim * sizeof(float),
                        hipMemcpyDeviceToHost));

  // 释放设备内存 (保留中间结果)
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_shared_w));
  CHECK_CUDA(hipFree(d_shared_b));
  CHECK_CUDA(hipFree(d_actor_fc1_w));
  CHECK_CUDA(hipFree(d_actor_fc1_b));
  CHECK_CUDA(hipFree(d_actor_fc2_w));
  CHECK_CUDA(hipFree(d_actor_fc2_b));
  CHECK_CUDA(hipFree(d_actor_head_w));
  CHECK_CUDA(hipFree(d_actor_head_b));
  CHECK_CUDA(hipFree(d_critic_fc1_w));
  CHECK_CUDA(hipFree(d_critic_fc1_b));
  CHECK_CUDA(hipFree(d_critic_fc2_w));
  CHECK_CUDA(hipFree(d_critic_fc2_b));
  CHECK_CUDA(hipFree(d_critic_head_w));
  CHECK_CUDA(hipFree(d_critic_head_b));

  // 销毁CUDA流
  CHECK_CUDA(hipStreamDestroy(actor_stream));
  CHECK_CUDA(hipStreamDestroy(critic_stream));
}

__global__ void merge_gradients_kernel(float *out, const float *a, const float *b, int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    out[idx] = a[idx] + b[idx];
  }
}

// 优化后的反向传播
void cuda_backward(const float *input, IntermediateData *fwd_data,
                   const float *grad_actor_output,
                   const float *grad_critic_output, int batch_size,
                   int input_dim, int hidden_dim, int output_dim,
                   const float *actor_fc2_w, const float *critic_fc2_w,
                   const float *actor_head_w, const float *critic_head_w,
                   float *grad_shared_w, float *grad_shared_b,
                   float *grad_actor_fc1_w, float *grad_actor_fc1_b,
                   float *grad_actor_fc2_w, float *grad_actor_fc2_b,
                   float *grad_actor_head_w, float *grad_actor_head_b,
                   float *grad_critic_fc1_w, float *grad_critic_fc1_b,
                   float *grad_critic_fc2_w, float *grad_critic_fc2_b,
                   float *grad_critic_head_w, float *grad_critic_head_b) {

  // 设备内存指针
  float *d_input, *d_actor_fc2_w, *d_critic_fc2_w;
  float *d_actor_head_w, *d_critic_head_w;
  float *d_grad_actor_output, *d_grad_critic_output;
  float *d_grad_shared_w, *d_grad_shared_b;
  float *d_grad_actor_fc1_w, *d_grad_actor_fc1_b;
  float *d_grad_actor_fc2_w, *d_grad_actor_fc2_b;
  float *d_grad_actor_head_w, *d_grad_actor_head_b;
  float *d_grad_critic_fc1_w, *d_grad_critic_fc1_b;
  float *d_grad_critic_fc2_w, *d_grad_critic_fc2_b;
  float *d_grad_critic_head_w, *d_grad_critic_head_b;

  // 梯度传递的中间结果
  float *d_grad_actor_fc2_output, *d_grad_critic_fc2_output;
  float *d_grad_actor_hidden, *d_grad_critic_hidden;
  float *d_grad_shared_output;

  // 分配设备内存
  CHECK_CUDA(hipMalloc(&d_input, batch_size * input_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_actor_fc2_w, output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_critic_fc2_w, output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_actor_head_w, output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_critic_head_w, output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_output,
                        batch_size * output_dim * sizeof(float)));

  CHECK_CUDA(
      hipMalloc(&d_grad_shared_w, hidden_dim * input_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_shared_b, hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_grad_actor_fc1_w, hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_fc1_b, hidden_dim * sizeof(float)));
  CHECK_CUDA(
      hipMalloc(&d_grad_actor_fc2_w, output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_fc2_b, output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_head_w,
                        output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_head_b, output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_fc1_w,
                        hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_fc1_b, hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_fc2_w,
                        output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_fc2_b, output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_head_w,
                        output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_head_b, output_dim * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_grad_actor_fc2_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_fc2_output,
                        batch_size * output_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_actor_hidden,
                        batch_size * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_critic_hidden,
                        batch_size * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_grad_shared_output,
                        batch_size * hidden_dim * sizeof(float)));

  // 拷贝数据到设备
  CHECK_CUDA(hipMemcpy(d_input, input, batch_size * input_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_fc2_w, actor_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_fc2_w, critic_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_actor_head_w, actor_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_critic_head_w, critic_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_grad_actor_output, grad_actor_output,
                        batch_size * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_grad_critic_output, grad_critic_output,
                        batch_size * output_dim * sizeof(float),
                        hipMemcpyHostToDevice));

  // 初始化梯度为0
  CHECK_CUDA(
      hipMemset(d_grad_shared_w, 0, hidden_dim * input_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_shared_b, 0, hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_fc1_w, 0,
                        hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_fc1_b, 0, hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_fc2_w, 0,
                        output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_fc2_b, 0, output_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_head_w, 0,
                        output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_actor_head_b, 0, output_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_fc1_w, 0,
                        hidden_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_fc1_b, 0, hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_fc2_w, 0,
                        output_dim * hidden_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_fc2_b, 0, output_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_head_w, 0,
                        output_dim * output_dim * sizeof(float)));
  CHECK_CUDA(hipMemset(d_grad_critic_head_b, 0, output_dim * sizeof(float)));

  // 创建CUDA流
  hipStream_t actor_stream, critic_stream;
  CHECK_CUDA(hipStreamCreate(&actor_stream));
  CHECK_CUDA(hipStreamCreate(&critic_stream));

  // Actor路径反向传播
  // 1. Actor Head层梯度
  dim3 grid_head_grad(output_dim, 1);
  dim3 block_head_grad(BLOCK_DIM);
  size_t shared_head_grad = BLOCK_DIM * sizeof(float);

  // 权重梯度
  optimized_weight_grad_kernel<<<grid_head_grad, block_head_grad,
                                 shared_head_grad, actor_stream>>>(
      fwd_data->d_actor_fc2_output, d_grad_actor_output, d_grad_actor_head_w,
      output_dim, output_dim, batch_size);

  // 偏置梯度
  optimized_bias_grad_kernel<<<grid_head_grad, block_head_grad,
                               shared_head_grad, actor_stream>>>(
      d_grad_actor_output, d_grad_actor_head_b, output_dim, batch_size);

  // 2. 计算Actor fc2输出梯度
  compute_hidden_grad_kernel<<<(batch_size * output_dim + BLOCK_DIM - 1) /
                                   BLOCK_DIM,
                               BLOCK_DIM, 0, actor_stream>>>(
      d_grad_actor_output, d_actor_head_w, d_grad_actor_fc2_output, output_dim,
      output_dim, batch_size);

  // 3. Actor fc2层梯度
  // 权重梯度
  optimized_weight_grad_kernel<<<dim3(output_dim, hidden_dim), block_head_grad,
                                 shared_head_grad, actor_stream>>>(
      fwd_data->d_actor_hidden, d_grad_actor_fc2_output, d_grad_actor_fc2_w,
      hidden_dim, output_dim, batch_size);

  // 偏置梯度
  optimized_bias_grad_kernel<<<dim3(output_dim, 1), block_head_grad,
                               shared_head_grad, actor_stream>>>(
      d_grad_actor_fc2_output, d_grad_actor_fc2_b, output_dim, batch_size);

  // 4. 计算Actor隐藏层梯度 (带tanh导数)
  compute_tanh_hidden_grad_kernel<<<(batch_size * hidden_dim + BLOCK_DIM - 1) /
                                        BLOCK_DIM,
                                    BLOCK_DIM, 0, actor_stream>>>(
      d_grad_actor_fc2_output, d_actor_fc2_w, fwd_data->d_actor_hidden,
      d_grad_actor_hidden, hidden_dim, output_dim, batch_size);

  // 5. Actor fc1层梯度
  // 权重梯度
  optimized_weight_grad_kernel<<<dim3(hidden_dim, hidden_dim), block_head_grad,
                                 shared_head_grad, actor_stream>>>(
      fwd_data->d_shared_output, d_grad_actor_hidden, d_grad_actor_fc1_w,
      hidden_dim, hidden_dim, batch_size);

  // 偏置梯度
  optimized_bias_grad_kernel<<<dim3(hidden_dim, 1), block_head_grad,
                               shared_head_grad, actor_stream>>>(
      d_grad_actor_hidden, d_grad_actor_fc1_b, hidden_dim, batch_size);

  // Critic路径反向传播 (与Actor类似)
  // 1. Critic Head层梯度
  optimized_weight_grad_kernel<<<grid_head_grad, block_head_grad,
                                 shared_head_grad, critic_stream>>>(
      fwd_data->d_critic_fc2_output, d_grad_critic_output, d_grad_critic_head_w,
      output_dim, output_dim, batch_size);
  optimized_bias_grad_kernel<<<grid_head_grad, block_head_grad,
                               shared_head_grad, critic_stream>>>(
      d_grad_critic_output, d_grad_critic_head_b, output_dim, batch_size);

  // 2. 计算Critic fc2输出梯度
  compute_hidden_grad_kernel<<<(batch_size * output_dim + BLOCK_DIM - 1) /
                                   BLOCK_DIM,
                               BLOCK_DIM, 0, critic_stream>>>(
      d_grad_critic_output, d_critic_head_w, d_grad_critic_fc2_output,
      output_dim, output_dim, batch_size);

  // 3. Critic fc2层梯度
  optimized_weight_grad_kernel<<<dim3(output_dim, hidden_dim), block_head_grad,
                                 shared_head_grad, critic_stream>>>(
      fwd_data->d_critic_hidden, d_grad_critic_fc2_output, d_grad_critic_fc2_w,
      hidden_dim, output_dim, batch_size);
  optimized_bias_grad_kernel<<<dim3(output_dim, 1), block_head_grad,
                               shared_head_grad, critic_stream>>>(
      d_grad_critic_fc2_output, d_grad_critic_fc2_b, output_dim, batch_size);

  // 4. 计算Critic隐藏层梯度 (带tanh导数)
  compute_tanh_hidden_grad_kernel<<<(batch_size * hidden_dim + BLOCK_DIM - 1) /
                                        BLOCK_DIM,
                                    BLOCK_DIM, 0, critic_stream>>>(
      d_grad_critic_fc2_output, d_critic_fc2_w, fwd_data->d_critic_hidden,
      d_grad_critic_hidden, hidden_dim, output_dim, batch_size);

  // 5. Critic fc1层梯度
  optimized_weight_grad_kernel<<<dim3(hidden_dim, hidden_dim), block_head_grad,
                                 shared_head_grad, critic_stream>>>(
      fwd_data->d_shared_output, d_grad_critic_hidden, d_grad_critic_fc1_w,
      hidden_dim, hidden_dim, batch_size);
  optimized_bias_grad_kernel<<<dim3(hidden_dim, 1), block_head_grad,
                               shared_head_grad, critic_stream>>>(
      d_grad_critic_hidden, d_grad_critic_fc1_b, hidden_dim, batch_size);

  // 等待两个流完成
  CHECK_CUDA(hipStreamSynchronize(actor_stream));
  CHECK_CUDA(hipStreamSynchronize(critic_stream));


  printf("Actor and Critic backward pass completed.\n");
  dim3 mergeBlock(256);
  dim3 mergeGrid(CEIL_DIV(batch_size * hidden_dim, mergeBlock.x));
  merge_gradients_kernel<<<mergeGrid, mergeBlock, 0, actor_stream>>>(
      d_grad_shared_output, 
      d_grad_actor_hidden, 
      d_grad_critic_hidden, 
      batch_size * hidden_dim
  );
  CHECK_CUDA(hipStreamSynchronize(actor_stream));
  // 合并共享层的梯度
  // for (int i = 0; i < batch_size * hidden_dim; i++) {
  //   d_grad_shared_output[i] = d_grad_actor_hidden[i] + d_grad_critic_hidden[i];
  // }
  printf("Shared layer gradients computed.\n");

  // 执行共享层的反向传播
  dim3 grid_shared_grad(hidden_dim, input_dim, 1);
  dim3 block_shared_grad(1, 1, 1);
  optimized_shared_backward_kernel<<<grid_shared_grad, block_shared_grad>>>(
      d_input, d_grad_actor_hidden, d_grad_critic_hidden, d_grad_shared_w,
      d_grad_shared_b, input_dim, hidden_dim, batch_size);

  // 拷贝梯度结果回主机
  CHECK_CUDA(hipMemcpy(grad_shared_w, d_grad_shared_w,
                        hidden_dim * input_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_shared_b, d_grad_shared_b,
                        hidden_dim * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipMemcpy(grad_actor_fc1_w, d_grad_actor_fc1_w,
                        hidden_dim * hidden_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_actor_fc1_b, d_grad_actor_fc1_b,
                        hidden_dim * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_actor_fc2_w, d_grad_actor_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_actor_fc2_b, d_grad_actor_fc2_b,
                        output_dim * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_actor_head_w, d_grad_actor_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_actor_head_b, d_grad_actor_head_b,
                        output_dim * sizeof(float), hipMemcpyDeviceToHost));

  CHECK_CUDA(hipMemcpy(grad_critic_fc1_w, d_grad_critic_fc1_w,
                        hidden_dim * hidden_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_critic_fc1_b, d_grad_critic_fc1_b,
                        hidden_dim * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_critic_fc2_w, d_grad_critic_fc2_w,
                        output_dim * hidden_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_critic_fc2_b, d_grad_critic_fc2_b,
                        output_dim * sizeof(float), hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_critic_head_w, d_grad_critic_head_w,
                        output_dim * output_dim * sizeof(float),
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipMemcpy(grad_critic_head_b, d_grad_critic_head_b,
                        output_dim * sizeof(float), hipMemcpyDeviceToHost));

  // 释放设备内存
  CHECK_CUDA(hipFree(d_input));
  CHECK_CUDA(hipFree(d_actor_fc2_w));
  CHECK_CUDA(hipFree(d_critic_fc2_w));
  CHECK_CUDA(hipFree(d_actor_head_w));
  CHECK_CUDA(hipFree(d_critic_head_w));
  CHECK_CUDA(hipFree(d_grad_actor_output));
  CHECK_CUDA(hipFree(d_grad_critic_output));
  CHECK_CUDA(hipFree(d_grad_shared_w));
  CHECK_CUDA(hipFree(d_grad_shared_b));
  CHECK_CUDA(hipFree(d_grad_actor_fc1_w));
  CHECK_CUDA(hipFree(d_grad_actor_fc1_b));
  CHECK_CUDA(hipFree(d_grad_actor_fc2_w));
  CHECK_CUDA(hipFree(d_grad_actor_fc2_b));
  CHECK_CUDA(hipFree(d_grad_actor_head_w));
  CHECK_CUDA(hipFree(d_grad_actor_head_b));
  CHECK_CUDA(hipFree(d_grad_critic_fc1_w));
  CHECK_CUDA(hipFree(d_grad_critic_fc1_b));
  CHECK_CUDA(hipFree(d_grad_critic_fc2_w));
  CHECK_CUDA(hipFree(d_grad_critic_fc2_b));
  CHECK_CUDA(hipFree(d_grad_critic_head_w));
  CHECK_CUDA(hipFree(d_grad_critic_head_b));
  CHECK_CUDA(hipFree(d_grad_actor_fc2_output));
  CHECK_CUDA(hipFree(d_grad_critic_fc2_output));
  CHECK_CUDA(hipFree(d_grad_actor_hidden));
  CHECK_CUDA(hipFree(d_grad_critic_hidden));
  CHECK_CUDA(hipFree(d_grad_shared_output));

  // 销毁CUDA流
  CHECK_CUDA(hipStreamDestroy(actor_stream));
  CHECK_CUDA(hipStreamDestroy(critic_stream));
}
} // extern "C"